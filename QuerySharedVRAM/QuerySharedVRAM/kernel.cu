
#include "hip/hip_runtime.h"
#include ""

#include <Windows.h>
#include <dxgi1_4.h>
#include <CL/cl.h>

#include <cstdio>

int main(int argc, const char *argv[])
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    auto cudaStatus = hipSetDevice(0);
    IDXGIFactory4* factory = nullptr;

    do
    {
        if (cudaStatus != hipSuccess)
        {
            puts("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
            break;
        }

        if (CreateDXGIFactory1(__uuidof(IDXGIFactory4), (void**)&factory) < 0)
            break;

        // Here, we can use a warp device
        IDXGIAdapter* warpAdapter;
        if (factory->EnumWarpAdapter(__uuidof(IDXGIAdapter), (void**)&warpAdapter) < 0)
            break;

        DXGI_ADAPTER_DESC desc;
        warpAdapter->GetDesc(&desc);

        // Here, we can choose which device to use
        IDXGIAdapter1* adapter1;
        if (factory->EnumAdapters1(0, &adapter1) < 0)
            break;

        DXGI_ADAPTER_DESC1 desc1;
        adapter1->GetDesc1(&desc1);

        constexpr auto oneGB = 1024.0 * 1024.0 * 1024.0;

        wprintf(L"Current GPU: %s\n", desc1.Description);
        printf("Current available total VRAM size: %.2fGB\n", (double)desc1.DedicatedVideoMemory / oneGB);
        printf("Current available total shared VRAM size: %.2fGB\n", (double)desc1.SharedSystemMemory / oneGB);

        size_t freeVRAM = 0, totalVRAM = 0;
        cudaStatus = hipMemGetInfo(&freeVRAM, &totalVRAM);
        if (cudaStatus == hipSuccess)
            printf("Queried by CUDA, free VRAM: %.2fGB, total: %.2fGB\n", (double)freeVRAM / oneGB, (double)totalVRAM / oneGB);

        cl_platform_id platformID = nullptr;
        auto clState = clGetPlatformIDs(1, &platformID, nullptr);
        if (clState != CL_SUCCESS || platformID == nullptr)
            break;

        cl_device_id deviceID = nullptr;
        clState = clGetDeviceIDs(platformID, CL_DEVICE_TYPE_GPU, 1, &deviceID, nullptr);
        if (clState != CL_SUCCESS || deviceID == nullptr)
            break;

        size_t vramSize = 0;
        clState = clGetDeviceInfo(deviceID, CL_DEVICE_GLOBAL_MEM_SIZE, sizeof(vramSize), &vramSize, nullptr);
        if (clState == CL_SUCCESS)
            printf("VRAM size queried by OpenCL: %.2fGB\n", (double)vramSize / oneGB);

        // Test virtual memory allocation
        constexpr auto bufferSize = 2ULL * (size_t)oneGB;

        void* vMem = VirtualAlloc(
            NULL,                       // System select
            bufferSize,                 // buffer size
            MEM_RESERVE | MEM_COMMIT,   // allocate reserved pages
            PAGE_READWRITE              // protection = read/write
        );

        if (vMem != nullptr)
        {
            constexpr auto nLoops = bufferSize / sizeof(unsigned);
            unsigned* buf = (unsigned*)vMem;
            for (size_t i = 0; i < nLoops; i++)
                buf[i] = (unsigned)i;

            VirtualFree(vMem, 0, MEM_RELEASE);
        }

    } while (false);

    if(factory != nullptr)
        factory->Release();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

